
#include <hip/hip_runtime.h>
__global__ void add(float* a, float* b, float* c, size_t n) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}