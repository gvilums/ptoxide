
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define BLOCK_SIZE 32 

__global__ void transpose(float *input, float *output, size_t N) {

	__shared__ float sharedMemory [BLOCK_SIZE] [BLOCK_SIZE];

	// global index	
	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;

	// transposed global memory index
	int tindexX = threadIdx.x + blockIdx.y * blockDim.x;
	int tindexY = threadIdx.y + blockIdx.x * blockDim.y;

	// local index
	int localIndexX = threadIdx.x;
	int localIndexY = threadIdx.y;

	int index = indexY * N + indexX;
	int transposedIndex = tindexY * N + tindexX;

	// reading from global memory in coalesed manner and performing tanspose in shared memory
	sharedMemory[localIndexX][localIndexY] = input[index];

	__syncthreads();

	// writing into global memory in coalesed fashion via transposed data in shared memory
	output[transposedIndex] = sharedMemory[localIndexY][localIndexX];
}
